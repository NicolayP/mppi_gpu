#include "hip/hip_runtime.h"
#include "gpu_cmake.hpp"
#include <iostream>
#include <assert.h>

__global__ void kernel(float* o, float* a, float* b, int n){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(tid < n){
    o[tid] = a[tid] + b[tid];
  }
}

int main(){
  float *h_o, *h_a, *h_b, *d_o, *d_a, *d_b;
  size_t bytes = sizeof(float)*N;

  int block_size = 256;
  int grid_size = ((N + block_size)/block_size);

  h_o = (float*) malloc(bytes);
  h_a = (float*) malloc(bytes);
  h_b = (float*) malloc(bytes);

  hipMalloc((void**)&d_o, bytes);
  hipMalloc((void**)&d_a, bytes);
  hipMalloc((void**)&d_b, bytes);

  for(int i=0; i < N; i++){
    h_a[i] = 1.0f;
    h_b[i] = 2.0f;
  }

  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  kernel<<<grid_size,block_size>>>(d_o, d_a, d_b, N);

  hipMemcpy(h_o, d_o, bytes, hipMemcpyDeviceToHost);

  for (int i=0; i < N; i++){
      assert(fabs(h_o[i] - h_a[i] - h_b[i]) < ERR_TOL);
  }
  std::cout << "Test passed" << std::endl;

  hipFree(d_o);
  hipFree(d_a);
  hipFree(d_b);
  free(h_o);
  free(h_a);
  free(h_b);

  return 0;
}
