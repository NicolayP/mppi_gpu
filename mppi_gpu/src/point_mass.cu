#include "hip/hip_runtime.h"
#include "point_mass.hpp"
#include <iostream>
/*__global__ void kernel(Model* model, int n){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  model->sim_gpu<<< 1 + n/256, 256>>>();
  hipDeviceSynchronize();
}*/


__host__ __device__ PointMassModelGpu::PointMassModelGpu(){
  x_ = nullptr;
  u_ = nullptr;
  tau_ = STEPS;
  u_gain_[0] = dt_*dt_/2.0;
  u_gain_[1] = dt_;
  x_gain_[0] = 1;
  x_gain_[1] = dt_;
  x_gain_[2] = 0;
  x_gain_[3] = 1;
  t_ = 1;

}

__host__ __device__ void PointMassModelGpu::init(float* x,
                                                 float init,
                                                 float* u,
                                                 float* x_gain,
                                                 int x_size,
                                                 float* u_gain,
                                                 int u_size){
  // TODO: cache x* in sm memory for faster access
  x_ = x;
  u_ = u;
  x_[0] = init;
  tau_ = STEPS;
  // Point the gain pointers to the right address
  x_gain_ = x_gain;
  x_size_ = x_size;

  u_gain_ = u_gain;
  u_size_ = u_size;

  t_ = 1;
  return;
}

__host__ __device__ void PointMassModelGpu::step(){
  for(int i=0; i < 2; i++){
    x_[t_*x_size_+i] = x_gain_[0]*x_[(t_-1)*x_size_+i] +
           x_gain_[1]*x_[(t_-1)*x_size_+i+1] +
           u_gain_[0]*u_[(t_-1)*u_size_ + i];

    x_[t_*x_size_+i+2] = x_gain_[2]*x_[(t_-1)*x_size_+i] +
             x_gain_[3]*x_[(t_-1)*x_size_+i+1] +
             u_gain_[1]*u_[(t_-1)*x_size_ + i];
  }
}

__host__ __device__ void PointMassModelGpu::run(){
  for (t_ = 1; t_ < tau_; t_++ ){
    step();
  }
}

__host__ __device__ void PointMassModelGpu::set_state(float* x){
  x_ = x;
}

__host__ __device__ void PointMassModelGpu::set_horizon(int horizon){
  /*
   * DO NOT USE ATM, when steps change, we need to update
   * the pointer x for the extra allocate space. As all the data
   * is represented in a continous array failling to do so will
   * produce a seg fault and probably leave the memory in a inconsistant
   * state.
   */
  tau_ = horizon;
}

__host__ __device__ float* PointMassModelGpu::get_state(){ return x_;}

__host__ __device__ int PointMassModelGpu::get_horizon(){ return tau_;}


PointMassModel::PointMassModel(int nb_sim, int steps, float dt){
  n_sim_ = nb_sim;
  steps_ = steps;
  act_dim = 2;
  state_dim = 4;

  dt_ = dt;


  /*
   * just for convinience, ultimatly replace with a
   * template type associated with the class wich will
   * represent the mppi domain.
   */
  bytes_ = sizeof(int)*steps_*n_sim_*state_dim;

  //host data used to send data to memory.
  float state_[4];
  float act_[2];

  act_[0] = dt_*dt_/2.0;
  act_[1] = dt_;
  state_[0] = 1;
  state_[1] = dt_;
  state_[2] = 0;
  state_[3] = 1;

  // *Allocate the data on tahe GPU.*

  // allocate space for all our simulation objects.
  hipMalloc((void**)&d_models, sizeof(PointMassModelGpu)*n_sim_);
  // allocate space for the init_state array. int* x[n_sim]
  hipMalloc((void**)&d_x_i, sizeof(float)*n_sim_*state_dim);
  // allocate data space, continous in memeory so int* x[n_sim*steps_]
  hipMalloc((void**)&d_x, sizeof(float)*n_sim_*steps_*state_dim);
  // set the memory with 0s.
  hipMemset((void*)d_x, 0, sizeof(float)*n_sim_*steps_*state_dim);
  // allocate space for action.
  hipMalloc((void**)&d_u, sizeof(float)*n_sim_*steps_*act_dim);

  // Set gain memory
  hipMalloc((void**)&state_gain, sizeof(float)*state_dim);
  hipMalloc((void**)&act_gain, sizeof(float)*act_dim);

  hipMemcpy(state_gain, state_, sizeof(float)*state_dim, hipMemcpyHostToDevice);
  hipMemcpy(act_gain, act_, sizeof(float)*act_dim, hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  std::cout << "Simulation objects created" << std::endl;
}

PointMassModel::~PointMassModel(){
  hipFree(d_x);
  hipFree(d_x_i);
  hipFree(d_models);
  hipFree(state_gain);
  hipFree(act_gain);
}

void PointMassModel::sim(){
  // launch 1 thread per simulation. Can later consider to
  // add dimensions to the block and thread of the kernel
  // to // enven more the code inside the simulation.
  // using blockDim.y & blockDim.z, blockIdx.y & blockIdx.x
  // and threadIdx.y & threadIdx.z.
  std::cout << "Starting simulations..." << std::endl;
  sim_gpu_kernel_<<<1 + n_sim_/256, 256>>>(d_models, n_sim_);
  std::cout << "simulations finished!" << std::endl;
  hipDeviceSynchronize();
}

void PointMassModel::memcpy_set_data(float* x, float* u){
  hipMemcpy(d_x_i, x, sizeof(float)*n_sim_*state_dim, hipMemcpyHostToDevice);
  hipMemcpy(d_u, u, sizeof(float)*n_sim_*act_dim*steps_, hipMemcpyHostToDevice);
  std::cout << "Setting inital state of the sims... : ";
  set_data_<<<1 + n_sim_/256, 256>>>(d_models,
                                     d_x_i,
                                     d_x,
                                     d_u,
                                     n_sim_,
                                     steps_,
                                     state_gain,
                                     state_dim,
                                     act_gain,
                                     act_dim);
  std::cout << "Done" << std::endl;
  hipDeviceSynchronize();
}

void PointMassModel::memcpy_get_data(float* x_all){
  hipMemcpy(x_all, d_x, bytes_, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}

__global__ void sim_gpu_kernel_(PointMassModelGpu* d_models, int n_sim_){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(tid < n_sim_){
    d_models[tid].run();
  }
  // sync thread.

  // Find min on the thread

  // get total min

  // compute normalisation term.

  // compute weight.

  // update actions

  // slide actions.
}

__global__ void set_data_(PointMassModelGpu* d_models,
                          float* d_x_i,
                          float* d_x,
                          float* d_u,
                          int n_sim,
                          int steps,
                          float* state_gain,
                          int state_dim,
                          float* act_gain,
                          int act_dim){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(tid < n_sim){
    d_models[tid].init(&d_x[tid*steps*state_dim], d_x_i[tid], &d_u[tid*steps*act_dim], state_gain, 4, act_gain, 2);
  }
}
