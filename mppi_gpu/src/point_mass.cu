#include "hip/hip_runtime.h"
#include "point_mass.hpp"

#include <iostream>
#include <math.h>

#define SIZE 256
#define SHMEM_SIZE 256


__host__ __device__ PointMassModelGpu::PointMassModelGpu(){
    _x = nullptr;
    _u = nullptr;
    _e = nullptr;
    _tau = STEPS;
    _t = 1;
    /*
    // Local copy of the weight and the goal.
    float* w = (float*) malloc(sizeof(float)*x_size);
    float* g = (float*) malloc(sizeof(float)*x_size);
    for (int i = 0; i < x_size; i++)
    {
        w[i] = weight[i];
        g[i] = goal[i];
    }
    _cost = Cost(w, x_size, g, x_size, lambda);
    _c = 0;*/
}

__host__ __device__ void PointMassModelGpu::init(float* x,
                                                 float init,
                                                 float* u,
                                                 float* e,
                                                 float* x_gain,
                                                 int x_size,
                                                 float* u_gain,
                                                 int u_size,
                                                 float* weight,
                                                 float* goal,
                                                 float lambda)
{
    // TODO: cache x* in sm memory for faster access
    _x = x;
    _u = u;
    _x[0] = init;
    _tau = STEPS;
    // Point the gain pointers to the right address
    _x_gain = x_gain;
    _x_size = x_size;

    _u_gain = u_gain;
    _u_size = u_size;

    _t = 1;
    // Local copy of the weight and the goal.
    _w = (float*) malloc(sizeof(float)*_x_size);
    _g = (float*) malloc(sizeof(float)*_x_size);
    // local copy of the error for faster access
    _e = (float*) malloc(sizeof(float)*STEPS*_u_size);

    _inv_s = (float*) malloc(sizeof(float)*_u_size);
    _inv_s[0] = 1.0;
    _inv_s[1] = 1.0;

    for (int i = 0; i < _x_size; i++)
    {
        _w[i] = weight[i];
        _g[i] = goal[i];
    }
    for (int i = 0; i < STEPS; i++){
        _e[i*_u_size + 0] = e[i*_u_size + 0];
        _e[i*_u_size + 1] = e[i*_u_size + 1];
    }
    _cost = Cost::Cost(_w, _x_size, _g, _x_size, lambda, _inv_s, u_size);
    _c = 0;
    return;
}

__host__ __device__ void PointMassModelGpu::step(hiprandState* state)
{

#ifdef __CUDA_ARCH__
    _e[(_t-1)*_u_size] += hiprand_normal(state);
    _e[(_t-1)*_u_size + 1] += hiprand_normal(state);
#else
    _e[(_t-1)*_u_size] += 0; //cpu random uniform;
    _e[(_t-1)*_u_size + 1] += 0; //cpu random uniform;
#endif
    for(int i=0; i < 2; i++){
        _x[_t*_x_size+i] = _x_gain[0]*_x[(_t-1)*_x_size+i] +
        _x_gain[1]*_x[(_t-1)*_x_size+i+2] +
        _u_gain[0]*(_u[(_t-1)*_u_size + i] + _e[(_t-1)*_u_size + i]);

        _x[_t*_x_size+i+2] = _x_gain[2]*_x[(_t-1)*_x_size+i] +
        _x_gain[3]*_x[(_t-1)*_x_size+i+2] +
        _u_gain[1]*(_u[(_t-1)*_u_size + i] + _e[(_t-1)*_u_size + i]);
    }
    _c += _cost.step_cost(&_x[_t*_x_size], &_u[(_t-1)*_u_size], &_e[(_t-1)*_u_size]);
}

__host__ __device__ float PointMassModelGpu::run(hiprandState* state)
{
    for (_t = 1; _t < _tau; _t++ ){
        step(state);
    }
    _c += _cost.final_cost(&_x[_t*_x_size]);
    return _c;
}

__host__ __device__ void PointMassModelGpu::set_state(float* x)
{
    _x = x;
}

__host__ __device__ void PointMassModelGpu::set_horizon(int horizon)
{
    /*
    * DO NOT USE ATM, when steps change, we need to update
    * the pointer x for the extra allocate space. As all the data
    * is represented in a continous array failling to do so will
    * produce a seg fault and probably leave the memory in a inconsistant
    * state.
    */
    _tau = horizon;
}

__host__ __device__ float* PointMassModelGpu::get_state(){ return _x;}

__host__ __device__ int PointMassModelGpu::get_horizon(){ return _tau;}


PointMassModel::PointMassModel(size_t nb_sim, size_t steps, float dt)
{
    n_sim_ = nb_sim;
    steps_ = steps;
    act_dim = 2;
    state_dim = 4;

    _dt = dt;


    /*
    * just for convinience, ultimatly replace with a
    * template type associated with the class wich will
    * represent the mppi domain.
    */
    bytes_ = sizeof(float)*steps_*n_sim_*state_dim;

    //host data used to send data to memory.
    float state_[4];
    float act_[2];

    act_[0] = _dt*_dt/2.0;
    act_[1] = _dt;
    state_[0] = 1;
    state_[1] = _dt;
    state_[2] = 0;
    state_[3] = 1;

    float lambda[0];
    lambda[0] = 1.;

    size_t GRID_SIZE = n_sim_ / SIZE / 2 + 1;
    // *Allocate the data on tahe GPU.*
    std::cout << "Allocating Space... : " << std::flush;
    // allocate space for all our simulation objects.
    CUDA_CALL_CONST(hipMalloc((void**)&d_models, sizeof(PointMassModelGpu)*n_sim_));
    // allocate space for the init_state array. int* x[n_sim]
    CUDA_CALL_CONST(hipMalloc((void**)&d_x_i, sizeof(float)*n_sim_*state_dim));
    // allocate data space, continous in memeory so int* x[n_sim*steps_]
    CUDA_CALL_CONST(hipMalloc((void**)&d_x, sizeof(float)*n_sim_*steps_*state_dim));
    // set the memory with 0s.
    CUDA_CALL_CONST(hipMemset((void*)d_x, 0, sizeof(float)*n_sim_*steps_*state_dim));
    // allocate space for action.
    CUDA_CALL_CONST(hipMalloc((void**)&d_e, sizeof(float)*n_sim_*steps_*act_dim));

    CUDA_CALL_CONST(hipMemset((void*)d_e, 0, sizeof(float)*n_sim_*steps_*act_dim));

    CUDA_CALL_CONST(hipMalloc((void**)&d_u, sizeof(float)*steps_*act_dim));

    CUDA_CALL_CONST(hipMalloc((void**)&d_cost, sizeof(float)*n_sim_));

    // container for the min value and the normalisation term.
    CUDA_CALL_CONST(hipMalloc((void**)&d_beta, sizeof(float)));
    // used for the reduction algorithm
    CUDA_CALL_CONST(hipMalloc((void**)&_d_beta, sizeof(float)*GRID_SIZE));
    CUDA_CALL_CONST(hipMemset((void*)_d_beta, 0, sizeof(float)*GRID_SIZE));

    CUDA_CALL_CONST(hipMalloc((void**)&d_nabla, sizeof(float)));
    // used for the reduction algorithm
    CUDA_CALL_CONST(hipMalloc((void**)&_d_nabla, sizeof(float)*GRID_SIZE));
    CUDA_CALL_CONST(hipMemset((void*)_d_nabla, 0, sizeof(float)*GRID_SIZE));

    CUDA_CALL_CONST(hipMalloc((void**)&d_lambda, sizeof(float)));

    CUDA_CALL_CONST(hipMalloc((void**)&d_weights, sizeof(float)*n_sim_));

    CUDA_CALL_CONST(hipMemcpy(d_lambda, lambda, sizeof(float), hipMemcpyHostToDevice));


    // Set gain memory
    CUDA_CALL_CONST(hipMalloc((void**)&state_gain, sizeof(float)*state_dim));
    CUDA_CALL_CONST(hipMalloc((void**)&act_gain, sizeof(float)*act_dim));

    CUDA_CALL_CONST(hipMemcpy(state_gain, state_, sizeof(float)*state_dim, hipMemcpyHostToDevice));
    CUDA_CALL_CONST(hipMemcpy(act_gain, act_, sizeof(float)*act_dim, hipMemcpyHostToDevice));

    CUDA_CALL_CONST(hipMalloc((void**)&rng_states, sizeof(hiprandState_t)*n_sim_));

    CUDA_CALL_CONST(hipMalloc((void**)&d_g, sizeof(float)*state_dim));
    CUDA_CALL_CONST(hipMalloc((void**)&d_w, sizeof(float)*state_dim));

    hipDeviceSynchronize();
    std::cout << "Done" << std::endl;
}

PointMassModel::~PointMassModel()
{
    CUDA_CALL_CONST(hipFree((void*)d_x));
    CUDA_CALL_CONST(hipFree((void*)d_x_i));
    CUDA_CALL_CONST(hipFree((void*)d_u));
    CUDA_CALL_CONST(hipFree((void*)d_e));
    CUDA_CALL_CONST(hipFree((void*)d_beta));
    CUDA_CALL_CONST(hipFree((void*)_d_beta));
    CUDA_CALL_CONST(hipFree((void*)d_nabla));
    CUDA_CALL_CONST(hipFree((void*)_d_nabla));
    CUDA_CALL_CONST(hipFree((void*)state_gain));
    CUDA_CALL_CONST(hipFree((void*)act_gain));
    CUDA_CALL_CONST(hipFree((void*)d_models));
    CUDA_CALL_CONST(hipFree((void*)rng_states));
}

void PointMassModel::sim()
{
    // launch 1 thread per simulation. Can later consider to
    // add dimensions to the block and thread of the kernel
    // to // enven more the code inside the simulation.
    // using blockDim.y & blockDim.z, blockIdx.y & blockIdx.x
    // and threadIdx.y & threadIdx.z.
    std::cout << "Running simulations... : " << std::flush;
    sim_gpu_kernel_<<<1 + n_sim_/SIZE, SIZE>>>(d_models, n_sim_, d_e, d_cost, rng_states);
    std::cout << "Done" << std::endl;

    // find min cost
    std::cout << "Compute min cost... : " << std::flush;
    min_beta();
    std::cout << "Done" << std::endl;

    std::cout << "Compute nabla... : " << std::flush;
    nabla();
    std::cout << "Done" << std::endl;

    //compute weights
    std::cout << "Compute weights... : " << std::flush;
    weights();
    std::cout << "Done" << std::endl;
    //weight<<<>>>();
    // compute new set of actions.
    //action<<<>>>();
    hipDeviceSynchronize();
}

void PointMassModel::memcpy_set_data(float* x, float* u, float* goal, float* w)
{
    CUDA_CALL_CONST(hipMemcpy(d_x_i, x, sizeof(float)*n_sim_*state_dim, hipMemcpyHostToDevice));
    CUDA_CALL_CONST(hipMemcpy(d_u, u, sizeof(float)*n_sim_*act_dim*steps_, hipMemcpyHostToDevice));
    CUDA_CALL_CONST(hipMemcpy(d_g, goal, sizeof(float)*state_dim, hipMemcpyHostToDevice));
    CUDA_CALL_CONST(hipMemcpy(d_w, w, sizeof(float)*state_dim, hipMemcpyHostToDevice));
    std::cout << "Setting inital state of the sims... : " << std::flush;
    set_data_<<<1 + n_sim_/256, 256>>>(d_models,
                                        d_x_i,
                                        d_x,
                                        d_u,
                                        d_e,
                                        n_sim_,
                                        steps_,
                                        state_gain,
                                        state_dim,
                                        act_gain,
                                        act_dim,
                                        rng_states,
                                        d_g,
                                        d_w,
                                        d_lambda);
    std::cout << "Done" << std::endl;
    hipDeviceSynchronize();
}

void PointMassModel::memcpy_get_data(float* x_all, float* e)
{
    CUDA_CALL_CONST(hipMemcpy(x_all, d_x, bytes_, hipMemcpyDeviceToHost));
    CUDA_CALL_CONST(hipMemcpy(e, d_e, sizeof(float)*n_sim_*act_dim*steps_, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
}

void PointMassModel::get_inf()
{
    // get all the info to look at the results and debug if necessary.
}

void PointMassModel::min_beta()
{
    size_t _n_sim(n_sim_);
    // TB Size
    int BLOCK_SIZE = SIZE;

    // Grid Size (cut in half) (No padding)
    int GRID_SIZE = _n_sim / BLOCK_SIZE / 2 + 1;

    // THIS shouldn't change size during the controller iterations.
    // should verify this and then allocate data in the init to improve
    // computation time


    if (GRID_SIZE == 1)
    {
        min_red << <1, BLOCK_SIZE >> > (d_cost, _d_beta, _n_sim);
    }
    else
    {
        // insure at least one pass.
        min_red << <GRID_SIZE, BLOCK_SIZE >> > (d_cost, _d_beta, _n_sim);

        _n_sim = GRID_SIZE;
        GRID_SIZE = _n_sim / BLOCK_SIZE / 2 + 1 ;

        while (GRID_SIZE - 1 > 1 )
        {
            min_red << <GRID_SIZE, BLOCK_SIZE >> > (_d_beta, _d_beta, _n_sim);
            _n_sim = GRID_SIZE;
            GRID_SIZE = _n_sim / BLOCK_SIZE / 2 + 1 ;
        }
        min_red << <1, BLOCK_SIZE >> > (_d_beta, _d_beta, _n_sim);
    }

    CUDA_CALL_CONST(hipMemcpy(d_beta, _d_beta, sizeof(float), hipMemcpyDeviceToDevice));
}

void PointMassModel::nabla()
{
    size_t _n_sim(n_sim_);
    // TB Size
    int BLOCK_SIZE = SIZE;

    // Grid Size (cut in half) (No padding)
    int GRID_SIZE = _n_sim / BLOCK_SIZE / 2 + 1;


    if (GRID_SIZE == 1)
    {
        sum_red_exp << <1, BLOCK_SIZE >> > (d_cost, d_lambda, d_beta, _d_nabla, _n_sim);
    }
    else
    {
        // insure at least one pass.
        sum_red_exp << <GRID_SIZE, BLOCK_SIZE >> > (d_cost, d_lambda, d_beta, _d_nabla, _n_sim);

        _n_sim = GRID_SIZE;
        GRID_SIZE = _n_sim / BLOCK_SIZE / 2 + 1 ;

        while (GRID_SIZE - 1 > 1 )
        {
            sum_red << <GRID_SIZE, BLOCK_SIZE >> > (_d_nabla, _d_nabla, _n_sim);
            _n_sim = GRID_SIZE;
            GRID_SIZE = _n_sim / BLOCK_SIZE / 2 + 1 ;
        }
        sum_red << <1, BLOCK_SIZE >> > (_d_nabla, _d_nabla, _n_sim);
    }
    CUDA_CALL_CONST(hipMemcpy(d_nabla, _d_nabla, sizeof(float), hipMemcpyDeviceToDevice));

}

void PointMassModel::weights()
{
    weights_kernel<<<1 + n_sim_/SIZE, SIZE>>>(d_cost, d_weights, d_lambda, d_beta, d_nabla, n_sim_);
}

void PointMassModel::update_act()
{
    for (size_t t=0; t < steps_; t++)
    {
        size_t _n_sim(n_sim_);
        // TB Size
        size_t BLOCK_SIZE = SIZE;

        // Grid Size (cut in half) (No padding)
        size_t GRID_SIZE = _n_sim / BLOCK_SIZE / 2 + 1;

        float* v_r;
        CUDA_CALL_CONST(hipMalloc((void**)v_r, sizeof(float)*GRID_SIZE*act_dim));


        if (GRID_SIZE == 1)
        {
            update_act_kernel << <1, BLOCK_SIZE >> > (v_r, d_weights, d_e, STEPS, t, act_dim, _n_sim);
        }
        else
        {
            // insure at least one pass.
            update_act_kernel << <GRID_SIZE, BLOCK_SIZE >> > (v_r, d_weights, d_e, STEPS, t, act_dim, _n_sim);

            _n_sim = GRID_SIZE;
            GRID_SIZE = _n_sim / BLOCK_SIZE / 2 + 1 ;

            while (GRID_SIZE - 1 > 1 )
            {
                sum_red << <GRID_SIZE, BLOCK_SIZE >> > (v_r, v_r, _n_sim);
                _n_sim = GRID_SIZE;
                GRID_SIZE = _n_sim / BLOCK_SIZE / 2 + 1 ;
            }
            sum_red << <1, BLOCK_SIZE >> > (v_r, v_r, _n_sim);
        }
        copy_act<<< 1, act_dim >>>(d_u, v_r, t, act_dim);

        CUDA_CALL_CONST(hipFree((void*)v_r));
    }
}

__global__ void sim_gpu_kernel_(PointMassModelGpu* d_models,
                                size_t n_sim_,
                                float* d_u,
                                float* cost,
                                hiprandState* rng_states)
{

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < n_sim_){
        /* local copy of rng state for faster generation. */
        hiprandState localState = rng_states[tid];
        cost[tid] = d_models[tid].run(&localState);
        //printf("tid: %d, cost[tid]: %f\n", tid, cost[tid]);
        /* copy back the state ohterwise the rng state is not working. */
        rng_states[tid] = localState;
        /*
        for (int i = 0; i < STEPS; i++){
            printf("%d, u_x: %f\n", tid*(STEPS*2) + i*2 + 0, d_u[tid*(STEPS*2) + i*2 + 0]);
            printf("%d, u_y: %f\n", tid*(STEPS*2) + i*2 + 1, d_u[tid*(STEPS*2) + i*2 + 1]);
        }
        */
    }
    // replace with a block sync threadrather than device.

    // Find min on the thread

    // get total min

    // compute normalisation term.

    // compute weight.

    // update actions

    // slide actions.
}

__global__ void min_red(float* v, float* beta, int n)
{
    // Allocate shared memory
	__shared__ float partial_min[SHMEM_SIZE];

	// Load elements AND do first add of reduction
	// Vector now 2x as long as number of threads, so scale i
	size_t i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	// Store first partial result instead of just the elements
    if (i + blockDim.x < n)
    {
        partial_min[threadIdx.x] = v[i] < v[i + blockDim.x] ? v[i] : v[i + blockDim.x];
    }
    else if (i < n)
    {
        partial_min[threadIdx.x] = v[i];
    }
    else
    {
        partial_min[threadIdx.x] = INFINITY;
    }
    //partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];

	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (size_t s = blockDim.x / 2; s > 0; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			partial_min[threadIdx.x] = partial_min[threadIdx.x] < partial_min[threadIdx.x + s] ? partial_min[threadIdx.x] : partial_min[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	if (threadIdx.x == 0) {
		beta[blockIdx.x] = partial_min[0];
        //printf("partial_min[0]: %f\n", partial_min[0]);
	}
}

__global__ void sum_red_exp(float* v, float* lambda_1, float* beta, float* v_r, int n)
{
    // Allocate shared memory
    __shared__ float partial_sum[SHMEM_SIZE];

    // Load elements AND do first add of reduction
    // Vector now 2x as long as number of threads, so scale i
    size_t i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    // Store first partial result instead of just the elements
    if (i + blockDim.x < n)
    {
       v[i] = expf(lambda_1[0] * (v[i] - beta[0]));
       v[i + blockDim.x] = expf(lambda_1[0] * (v[i + blockDim.x] - beta[0]));
       partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];
    }
    else if (i < n)
    {
       v[i] = expf(lambda_1[0] * (v[i] - beta[0]));
       partial_sum[threadIdx.x] = v[i];
    }
    else
    {
        partial_sum[threadIdx.x] = 0;
    }
    //partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];

    __syncthreads();

    // Start at 1/2 block stride and divide by two each iteration
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1) {
        // Each thread does work unless it is further than the stride
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Let the thread 0 for this block write it's result to main memory
    // Result is inexed by this block
    if (threadIdx.x == 0) {
        v_r[blockIdx.x] = partial_sum[0];
    }
}

__global__ void sum_red(float* v, float* v_r, int n)
{
    // Allocate shared memory
    __shared__ float partial_sum[SHMEM_SIZE];

    // Load elements AND do first add of reduction
    // Vector now 2x as long as number of threads, so scale i
    size_t i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    // Store first partial result instead of just the elements
    if (i + blockDim.x < n)
    {
       partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];
    }
    else if (i < n)
    {
       partial_sum[threadIdx.x] = v[i];
    }
    else
    {
        partial_sum[threadIdx.x] = 0;
    }
    //partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];

    __syncthreads();

    // Start at 1/2 block stride and divide by two each iteration
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1) {
        // Each thread does work unless it is further than the stride
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Let the thread 0 for this block write it's result to main memory
    // Result is inexed by this block
    if (threadIdx.x == 0) {
        v_r[blockIdx.x] = partial_sum[0];
    }
}

__global__ void weights_kernel(float* v, float* v_r, float* lambda_1, float* beta, float* nabla_1, size_t n)
{
    size_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n){
        v_r[tid] = nabla_1[0] * expf(-lambda_1[0]*(v[tid] - beta[0]));
    }
}

__global__ void copy_act(float* u, float* tmp, size_t t, size_t act_dim){
    size_t tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < act_dim)
        u[tid + t*act_dim] += tmp[tid];
}
// First implementation. Usually T << K so parallelize over K first.
__global__ void update_act_kernel(float* v_r,
                                  float* w,
                                  float* e,
                                  size_t steps,
                                  size_t t,
                                  const size_t act_dim,
                                  size_t n)
{
    // Allocate shared memory
    const size_t a_dim(act_dim);

    __shared__ float partial_acts[SHMEM_SIZE*2];

    // Load elements AND do first add of reduction
    // Vector now 2x as long as number of threads, so scale i
    size_t i = blockIdx.x * (blockDim.x * 2) * act_dim + threadIdx.x * steps + t * act_dim;
    size_t k = blockIdx.x * blockDim.x + threadIdx.x;
    // Store first partial result instead of just the elements
    if (i + blockDim.x*steps*act_dim < n)
    {
        for (size_t j = 0; j < act_dim; j++)
        {
            partial_acts[threadIdx.x*act_dim + j] = w[k]*e[i + j] +
                                                    w[k]*e[i + blockDim.x*steps*act_dim + j];
        }
    }
    else if (i < n*steps*act_dim )
    {
        for (size_t j = 0; j < act_dim; j++)
        {
            partial_acts[threadIdx.x*act_dim + j] = w[k]*e[i + j];
        }
    }
    else
    {
        for (size_t j = 0; j < act_dim; j++)
        {
            partial_acts[threadIdx.x*act_dim + j] = 0;
        }
    }
    //partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];

    __syncthreads();

    // Start at 1/2 block stride and divide by two each iteration
    for (size_t s = blockDim.x * act_dim / 2; s > 0; s >>= 1)
    {
        // Each thread does work unless it is further than the stride
        if (threadIdx.x < s)
        {
            for (size_t j = 0; j < act_dim; j++)
            {
                partial_acts[threadIdx.x*act_dim + j] += partial_acts[threadIdx.x*act_dim + j + s];
            }
        }
        __syncthreads();
    }

    // Let the thread 0 for this block write it's result to main memory
    // Result is inexed by this block
    if (threadIdx.x == 0)
    {
        for(size_t j = 0; j < act_dim; j++)
        {
            v_r[blockIdx.x*act_dim + j] = partial_acts[j];
        }
    }
}

__global__ void set_data_(PointMassModelGpu* d_models,
                          float* d_x_i,
                          float* d_x,
                          float* d_u,
                          float* d_e,
                          size_t n_sim,
                          size_t steps,
                          float* state_gain,
                          size_t state_dim,
                          float* act_gain,
                          size_t act_dim,
                          hiprandState* rng_states,
                          float* goal,
                          float* w,
                          float* lambda)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < n_sim){
        hiprand_init(tid, tid, tid, &rng_states[tid]);
        d_models[tid].init(&d_x[tid*steps*state_dim],
                            d_x_i[tid],
                            d_u,
                            &d_e[tid*steps*act_dim],
                            state_gain,
                            state_dim,
                            act_gain,
                            act_dim,
                            w,
                            goal,
                            lambda[0]);
    }
}
