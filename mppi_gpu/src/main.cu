#include "hip/hip_runtime.h"
#include "point_mass.hpp"
#include "mppi_env.hpp"
#include "mppi_utils.hpp"
#include <iostream>
#include <fstream>

#include <assert.h>

#include <cstdio>
#include <ctime>
#include <chrono>
#include <unistd.h>

#include <tclap/CmdLine.h>
#include <yaml-cpp/yaml.h>
/*
* Same as previous example only this time the parallized
* objects contain a pointer to the data. This will show
* us how to implment a recording of a path for MPPI.
*
* This examples run a simple linear simulation in parallel.
* All this is wrapped in a model class which will later be our
* controller. The Wrapper creates simulation classes on the device.
* It then sets the input data on the classes (can be run in // too).
* finally it runs the simulation in parallel and collects the data
* on the host device.
*/


void to_csv (std::string filename,
            float* x,
            float* u,
            int sample,
            int size,
            int s_dim,
            int a_dim) {
    std::cout << "Saving data to file...: " << std::flush;
    std::ofstream outfile;
    // create a name for the file output

    outfile.open(filename);

    outfile << "sample" << "," << "x" << "," << "y" << "," << "x_dot" << ","
            << "y_dot" << "," << "u_x" << "," << "u_y" << std::endl;
    for (int i=0; i < sample; i++){
        for (int j=0; j < size; j++){
            outfile << i << ","
                    << x[i*size*s_dim + j*s_dim + 0] << ","
                    << x[i*size*s_dim + j*s_dim + 1] << ","
                    << x[i*size*s_dim + j*s_dim + 2] << ","
                    << x[i*size*s_dim + j*s_dim + 3] << ","
                    << u[i*size*a_dim + j*a_dim + 0] << ","
                    << u[i*size*a_dim + j*a_dim + 1] << std::endl;
        }
    }
    outfile.close();
    std::cout << "Done" << std::endl;
    return;
}

void to_csv2 (std::string filename,
             float* x,
             float* u,
             float* u_prev,
             float* e,
             float* cost,
             float* beta,
             float* nabla,
             float* w,
             int sample,
             int size,
             int s_dim,
             int a_dim) {

    std::cout << "Saving data to file...: " << std::flush;
    std::ofstream outfile;
    // create a name for the file output

    outfile.open(filename);

    outfile << "sample" << "," << "x" << "," << "y" << "," << "x_dot" << ","
            << "y_dot" << "," << "e_x" << "," << "e_y";
    for(int d=0; d < a_dim; d++)
    {
        outfile << "," << "u[" << d << "]";
    }

    for(int d=0; d < a_dim; d++)
    {
        outfile << "," << "u_prev[" << d << "]";
    }

    outfile << "," << "c" <<  "," << "w" << std::endl;

    for (int i=0; i < sample; i++){
        for (int j=0; j < size + 1 ; j++){
            outfile << i << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 0] << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 1] << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 2] << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 3] << ",";
            if (j < size) {
                outfile << e[i*size*a_dim + j*a_dim + 0] << ","
                        << e[i*size*a_dim + j*a_dim + 1];
            } else {
                outfile << ", ";
            }
            // U is of size steps
            if(i < 1 && j < size) {
                outfile << "," << u[j*a_dim + 0] << "," << u[j*a_dim + 1];
                outfile << "," << u_prev[j*a_dim + 0] << "," << u_prev[j*a_dim + 1];
            }else
                outfile << ", , , , ";
            if (i*size + j < sample) {
                outfile << "," << cost[i*size+j] << "," << w[i*size+j];
            }
            outfile << std::endl;
        }
    }


    outfile.close();
    std::cout << "Done" << std::endl;
    return;
}

void parse_argument (int argc,
                     char const* argv[],
                     std::string& config,
                     std::string& mjkey,
                     std::string& outfile);

void parse_config (std::string& config_file,
                   std::string& model_file,
                   int& samples,
                   int& state_dim,
                   int& act_dim,
                   int& horizon,
                   float& dt,
                   float& lambda,
                   float** noise,
                   float** init,
                   float** max_a,
                   float** goal,
                   std::string& cost_type,
                   float** cost_q);

void init_controller_var (int const& samples,
                          int const& state_dim,
                          int const& act_dim,
                          int const& horizon,
                          float** next_act,
                          float** init_state,
                          float** init_actions,
                          float** cost,
                          float** beta,
                          float** nabla,
                          float** weight);

void free_controller_memory(float** next_act,
                            float** init_state,
                            float** init_actions,
                            float** cost,
                            float** beta,
                            float** nabla,
                            float** weight);

void free_paresd_data(float** noise,
                      float** init_next,
                      float** max_a,
                      float** goal,
                      float** cost_q);

void init_action_seq(float* init_actions, int act_dim, int steps);

void verify_parse (int n,
                   int state_dim,
                   int act_dim,
                   int steps,
                   float dt,
                   float lambda,
                   float* noise,
                   float* init_next,
                   float* max_a,
                   float* cost_q,
                   float* goal);

int main (int argc, char const* argv[]) {

    std::string config_file;
    std::string mjkey_file;
    std::string out_file;
    std::string model_file;
    std::string cost_type;
    // will store the next action.
    float* next_act;

    int n;
    int state_dim;
    int act_dim;
    int steps;
    float dt;
    float lambda;
    float* noise;
    float* init_next;
    float* max_a;
    float* cost_q;
    float* goal;

    float* init_state;
    float* init_actions;
    float* cost;
    float* beta;
    float* nabla;
    float* weight;

    bool save = true;
    bool done=false;


    parse_argument(argc, argv, config_file, mjkey_file, out_file);

    parse_config(config_file,
                 model_file,
                 n,
                 state_dim,
                 act_dim,
                 steps,
                 dt,
                 lambda,
                 &noise,
                 &init_next,
                 &max_a,
                 &goal,
                 cost_type,
                 &cost_q);

    init_controller_var(n,
                         state_dim,
                         act_dim,
                         steps,
                         &next_act,
                         &init_state,
                         &init_actions,
                         &cost,
                         &beta,
                         &nabla,
                         &weight);

    if (config_file == "../config/mppi-config-test.yaml") {
        verify_parse(n,
                 state_dim,
                 act_dim,
                 steps,
                 dt,
                 lambda,
                 noise,
                 init_next,
                 max_a,
                 cost_q,
                 goal);
    }

    PointMassEnv env = PointMassEnv(model_file.c_str(), mjkey_file.c_str(), true);

    PointMassModel* model = new PointMassModel(n, steps, dt, state_dim, act_dim, false);

    env.get_x(init_state);
    init_action_seq(init_actions, act_dim, steps);

    model->memcpy_set_data(init_state, init_actions, goal, cost_q);

    // run the multiple simulation on the device.
    float* u_prev = (float*) malloc(sizeof(float)*steps*act_dim);
    while(!done){
        model->get_u(u_prev);
        //t1 = std::chrono::system_clock::now();
        model->get_act(next_act);
        //t2 = std::chrono::system_clock::now();
        //fp_ms += t2 - t1;
        done = env.simulate(next_act);
        //std::cout << "next_act: " << next_act[0] << ", " << next_act[1] << '\n';
        env.get_x(init_state);
        model->set_x(init_state);
    }

    if(save){
        float* h_x = (float*) malloc(sizeof(float)*n*(steps+1)*state_dim);
        float* h_u = (float*) malloc(sizeof(float)*steps*act_dim);
        float* h_e = (float*) malloc(sizeof(float)*n*steps*act_dim);
        float* u_prev = (float*) malloc(sizeof(float)*steps*act_dim);

        model->get_inf(h_x, h_u, h_e, cost, beta, nabla, weight);
        to_csv2(out_file, h_x, h_u, u_prev, h_e, cost, beta, nabla, weight, n, steps, state_dim, act_dim);
        free(h_x);
        free(h_u);
        free(h_e);
    }
    free(u_prev);

    std::cout << "Freeing memory... : " << std::flush;
    free_controller_memory(&next_act,
                           &init_state,
                           &init_actions,
                           &cost,
                           &beta,
                           &nabla,
                           &weight);
    free_paresd_data(&noise, &init_next, &max_a, &goal, &cost_q);
    delete model;
    std::cout << "Done" << std::endl;

    hipDeviceReset();
}

void parse_argument (int argc,
                     char const* argv[],
                     std::string& config,
                     std::string& mjkey,
                     std::string& outfile) {
    try {

        TCLAP::CmdLine cmd("Mppi controller", ' ', "0.0");
        TCLAP::ValueArg<std::string> configArg("c",
                                               "config",
                                               "Config file",
                                               false,
                                               "../config/point_mass.yaml",
                                               "string",
                                               cmd);

        TCLAP::ValueArg<std::string> mjkeyArg("k",
                                              "key",
                                              "Mujoco key file",
                                              false,
                                              "../lib/contrib/mjkey.txt",
                                              "string",
                                              cmd);

        TCLAP::ValueArg<std::string> outArg("o",
                                            "out",
                                            "Outpute file",
                                            false,
                                            "to_plot.csv",
                                            "string",
                                            cmd);

        cmd.parse(argc, argv);

        config = configArg.getValue();
        mjkey = mjkeyArg.getValue();
        outfile = outArg.getValue();

        std::cout << "Argument parsed" << std::endl;

    } catch (TCLAP::ArgException &e) {
        std::cerr << "error: " << e.error() << " for arg " << e.argId() << std::endl;
    }
}

void parse_config (std::string& config_file,
                   std::string& model_file,
                   int& samples,
                   int& state_dim,
                   int& act_dim,
                   int& horizon,
                   float& dt,
                   float& lambda,
                   float** noise,
                   float** init,
                   float** max_a,
                   float** goal,
                   std::string& cost_type,
                   float** cost_q) {
    float* tmp_noise;
    float* tmp_init;
    float* tmp_max_a;
    float* tmp_goal;
    float* tmp_cost_q;

    YAML::Node config = YAML::LoadFile(config_file);

    /* env section */
    if (!config["env"])  {
        std::cout << "Please provide a env file in the config file" << std::endl;
        exit(1);
    }
    model_file = config["env"].as<std::string>();

    /* Sample section */
    if (!config["samples"])  {
        std::cout << "Please provide the number of samples in the config file" << std::endl;
        exit(1);
    }
    samples = config["samples"].as<int>();

    /* State section */
    if (!config["state-dim"])  {
        std::cout << "Please provide the state dimension in the config file" << std::endl;
        exit(1);
    }
    state_dim = config["state-dim"].as<int>();

    /* Action section */
    if (!config["action-dim"])  {
        std::cout << "Please provide the action dimension in the config file" << std::endl;
        exit(1);
    }
    act_dim = config["action-dim"].as<int>();

    /* Horizon section */
    if (!config["horizon"])  {
        std::cout << "Please provide the prediction horizon in the config file" << std::endl;
        exit(1);
    }
    horizon = config["horizon"].as<int>();

    /* Timestep section */
    if (!config["dt"])  {
        std::cout << "Please provide the time step in the config file" << std::endl;
        exit(1);
    }
    dt = config["dt"].as<float>();

    /* Lambda section */
    if (!config["lambda"])  {
        std::cout << "Please provide a env file in the config file" << std::endl;
        exit(1);
    }
    lambda = config["lambda"].as<float>();

    /* Noise section */
    {
        if (!config["noise"])  {
            std::cout << "Please provide a noise vector in the config file, should be a array of size action-dim" << std::endl;
            exit(1);
        }
        if (config["noise"].size() != act_dim) {
            std::cout << "Warning: the cost function weights matrix is larger than the action dimension ";
        }
        tmp_noise = (float*) malloc(sizeof(float)*config["noise"].size());
    }


    /* Init action section */
    {
        if (!config["init-act"])  {
            std::cout << "Please provide a init vector in the config file, should be a array of size action-dim" << std::endl;
            exit(1);
        }
        if (config["init-act"].size() != act_dim) {
            std::cout << "Warning: the cost function weights matrix is larger than the action dimension ";
        }
        tmp_init = (float*) malloc(sizeof(float)*config["max-a"].size());
    }


    /* Max action section */
    {
        if (!config["max-a"])  {
            std::cout << "Please provide a max input vector in the config file, should be a array of size action-dim" << std::endl;
            exit(1);
        }
        if (config["max-a"].size() != act_dim) {
            std::cout << "Warning: the input limit is different than the action dimension " << std::endl;
        }
        tmp_max_a = (float*) malloc(sizeof(float)*config["max-a"].size());
    }


    for (std::size_t i=0; i < config["max-a"].size(); i++) {
        tmp_noise[i] = config["noise"][i].as<float>();
        tmp_init[i] = config["init-act"][i].as<float>();
        tmp_max_a[i] = config["max-a"][i].as<float>();
    }

    /* Goal section */
    {
        if (!config["goal"])  {
            std::cout << "Please provide a goal vector in the config file, should be a array of size action-dim" << std::endl;
            exit(1);
        }
        if (config["goal"].size() != state_dim) {
            std::cout << "Warning: the goal size is different than the state dimension " << std::endl;
        }
        tmp_goal = (float*) malloc(sizeof(float)*config["goal"].size());

        for (std::size_t i=0; i < config["goal"].size(); i++) {
            tmp_goal[i] = config["goal"][i].as<float>();
        }
    }

    /* Cost related section  */
    {
        if (!config["cost"])  {
            std::cout << "Please provide cost function in the config file." << std::endl;
            exit(1);
        }

        if (!config["cost"]["type"]) {
            std::cout << "Please provide cost function type in the config file. Currently supported: quadratic " << std::endl;
            exit(1);
        }
        cost_type = config["cost"]["type"].as<std::string>();

        if (!config["cost"]["w"]) {
            std::cout << "Please provide cost function type in the config file. Currently supported: quadratic " << std::endl;
            exit(1);
        }
        if (config["cost"]["w"].size() != state_dim) {
            std::cout << "Warning: the cost function weights matrix is different than the state dimension " << std::endl;
        }
        tmp_cost_q = (float*) malloc(sizeof(float)*config["cost"]["w"].size());

        for (std::size_t i=0; i< config["cost"]["w"].size(); i++) {
            tmp_cost_q[i] = config["cost"]["w"][i].as<float>();
        }
    }

    *noise = tmp_noise;
    *init = tmp_init;
    *max_a = tmp_max_a;
    *goal = tmp_goal;
    *cost_q = tmp_cost_q;

    tmp_noise = nullptr;
    tmp_init = nullptr;
    tmp_max_a = nullptr;
    tmp_cost_q = nullptr;
    tmp_goal = nullptr;

    std::cout << "N " << samples << " STEPS: " << horizon << " State dim: " << state_dim << std::endl;

}

void init_controller_var (int const& samples,
                          int const& state_dim,
                          int const& act_dim,
                          int const& horizon,
                          float** next_act,
                          float** init_state,
                          float** init_actions,
                          float** cost,
                          float** beta,
                          float** nabla,
                          float** weight) {
      *next_act = (float*) malloc(sizeof(float)*act_dim);
      *init_state = (float*) malloc(sizeof(float)*state_dim);
      *init_actions = (float*) malloc(sizeof(float)*horizon*act_dim);
      *cost = (float*) malloc(sizeof(float)*samples);
      *beta = (float*) malloc(sizeof(float));
      *nabla = (float*) malloc(sizeof(float));
      *weight = (float*) malloc(sizeof(float)*samples);
}

void free_controller_memory(float** next_act,
                            float** init_state,
                            float** init_actions,
                            float** cost,
                            float** beta,
                            float** nabla,
                            float** weight) {
    free(*next_act);
    free(*init_state);
    free(*init_actions);
    free(*cost);
    free(*beta);
    free(*nabla);
    free(*weight);
}

void free_paresd_data(float** noise,
                      float** init_next,
                      float** max_a,
                      float** goal,
                      float** cost_q) {
    free(*noise);
    free(*init_next);
    free(*max_a);
    free(*goal);
    free(*cost_q);
}

void init_action_seq(float* init_actions, int action_dim, int steps) {
    for (int i=0; i < steps; i++) {
        for( int j=0; j < action_dim; j++) {
            init_actions[i*action_dim + j] = 0.;
        }
    }
}

void verify_parse (int n,
                   int state_dim,
                   int act_dim,
                   int steps,
                   float dt,
                   float lambda,
                   float* noise,
                   float* init_next,
                   float* max_a,
                   float* cost_q,
                   float* goal) {
    assert(n == 3);
    assert(state_dim == 4);
    assert(act_dim == 2);
    assert(steps == 12);

    assert(fabs(dt - 0.1) < TOL);
    assert(fabs(lambda - 1.5) < TOL);
    assert(fabs(max_a[0] - 1.2) < TOL);
    assert(fabs(max_a[1] - 1.3) < TOL);

    assert(fabs(noise[0] - 0.24) < TOL);
    assert(fabs(noise[1] - 0.26) < TOL);


    assert(fabs(init_next[0] - 0.1) < TOL);
    assert(fabs(init_next[1] - 0.2) < TOL);

    assert(fabs(cost_q[0] - 1) < TOL);
    assert(fabs(cost_q[1] - 2) < TOL);
    assert(fabs(cost_q[2] - 0.5) < TOL);
    assert(fabs(cost_q[3] - 0.75) < TOL);

    assert(fabs(goal[0] - 1) < TOL);
    assert(fabs(goal[1] - 2) < TOL);
    assert(fabs(goal[2] - 3) < TOL);
    assert(fabs(goal[3] - 4) < TOL);

    std::cout << "Test passed" << std::endl;
}
