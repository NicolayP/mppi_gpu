#include "hip/hip_runtime.h"
#include "point_mass.hpp"
#include "mppi_env.hpp"
#include "mppi_utils.hpp"
#include <iostream>
#include <fstream>

#include <assert.h>

#include <cstdio>
#include <ctime>
#include <chrono>
#include <unistd.h>

#include <tclap/CmdLine.h>
#include <yaml-cpp/yaml.h>
/*
* Same as previous example only this time the parallized
* objects contain a pointer to the data. This will show
* us how to implment a recording of a path for MPPI.
*
* This examples run a simple linear simulation in parallel.
* All this is wrapped in a model class which will later be our
* controller. The Wrapper creates simulation classes on the device.
* It then sets the input data on the classes (can be run in // too).
* finally it runs the simulation in parallel and collects the data
* on the host device.
*/


void to_csv (std::string filename,
            float* x,
            float* u,
            int sample,
            int size,
            int s_dim,
            int a_dim)
{
    std::cout << "Saving data to file...: " << std::flush;
    std::ofstream outfile;
    // create a name for the file output

    outfile.open(filename);

    outfile << "sample" << "," << "x" << "," << "y" << "," << "x_dot" << ","
            << "y_dot" << "," << "u_x" << "," << "u_y" << std::endl;
    for (int i=0; i < sample; i++){
        for (int j=0; j < size; j++){
            outfile << i << ","
                    << x[i*size*s_dim + j*s_dim + 0] << ","
                    << x[i*size*s_dim + j*s_dim + 1] << ","
                    << x[i*size*s_dim + j*s_dim + 2] << ","
                    << x[i*size*s_dim + j*s_dim + 3] << ","
                    << u[i*size*a_dim + j*a_dim + 0] << ","
                    << u[i*size*a_dim + j*a_dim + 1] << std::endl;
        }
    }
    outfile.close();
    std::cout << "Done" << std::endl;
    return;
}

void to_csv2 (std::string filename,
             float* x,
             float* u,
             float* u_prev,
             float* e,
             float* cost,
             float* beta,
             float* nabla,
             float* w,
             int sample,
             int size,
             int s_dim,
             int a_dim) {

    std::cout << "Saving data to file...: " << std::flush;
    std::ofstream outfile;
    // create a name for the file output

    outfile.open(filename);

    outfile << "sample" << "," << "x" << "," << "y" << "," << "x_dot" << ","
            << "y_dot" << "," << "e_x" << "," << "e_y";
    for(int d=0; d < a_dim; d++)
    {
        outfile << "," << "u[" << d << "]";
    }

    for(int d=0; d < a_dim; d++)
    {
        outfile << "," << "u_prev[" << d << "]";
    }

    outfile << "," << "c" <<  "," << "w" << std::endl;

    for (int i=0; i < sample; i++){
        for (int j=0; j < size + 1 ; j++){
            outfile << i << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 0] << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 1] << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 2] << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 3] << ",";
            if (j < size) {
                outfile << e[i*size*a_dim + j*a_dim + 0] << ","
                        << e[i*size*a_dim + j*a_dim + 1];
            } else {
                outfile << ", ";
            }
            // U is of size steps
            if(i < 1 && j < size) {
                outfile << "," << u[j*a_dim + 0] << "," << u[j*a_dim + 1];
                outfile << "," << u_prev[j*a_dim + 0] << "," << u_prev[j*a_dim + 1];
            }else
                outfile << ", , , , ";
            if (i*size + j < sample) {
                outfile << "," << cost[i*size+j] << "," << w[i*size+j];
            }
            outfile << std::endl;
        }
    }


    outfile.close();
    std::cout << "Done" << std::endl;
    return;
}

void parse_argument (int argc,
                     char const* argv[],
                     std::string& config,
                     std::string& mjkey,
                     std::string& outfile);

void parse_config (std::string& config_file,
                   std::string& model_file,
                   int& samples,
                   int& state_dim,
                   int& act_dim,
                   int& horizon,
                   float& lambda,
                   float** noise,
                   float** init,
                   float** max_a,
                   std::string& cost_type,
                   float** cost_q);

int main (int argc, char const* argv[]) {

    std::string config_file;
    std::string mjkeyFile;
    std::string outFile;
    std::string model_file;
    std::string cost_type;

    int n(0);
    int state_dim(0);
    int act_dim(0);
    int steps(0);
    float lambda;
    float* noise;
    float* init;
    float* max_a;
    float* cost_q;

    parse_argument(argc, argv, config_file, mjkeyFile, outFile);

    std::cout << "Config: " << config_file << std::endl;
    std::cout << "MjKey: " << mjkeyFile << std::endl;
    std::cout << "Outfile: " << outFile << std::endl;

    std::cout << max_a << std::endl;
    parse_config(config_file,
                 model_file,
                 n,
                 state_dim,
                 act_dim,
                 steps,
                 lambda,
                 &noise,
                 &init,
                 &max_a,
                 cost_type,
                 &cost_q);
    std::cout << max_a << std::endl;

    std::cout << "Parse config output: " << model_file << " "
              << n << " "
              << state_dim << " "
              << act_dim << " "
              << steps << " "
              << lambda << " " << std::endl;

    std::cout << "max_a: ";
    for (int i = 0; i < act_dim; i++) {
        std::cout << max_a[i] << ' ';
    }
    std::cout << std::endl;

    std::cout << "Init: ";
    for (int i = 0; i < act_dim; i++) {
        std::cout << init[i] << ' ';
    }
    std::cout << std::endl;

    std::cout << "Noise: ";
    for (int i = 0; i < act_dim; i++) {
        std::cout << noise[i] << ' ';
    }
    std::cout << std::endl;

    std::cout << "Cost_q: ";
    for (int i = 0; i < state_dim; i++) {
        std::cout << cost_q[i] << ' ';
    }
    std::cout << std::endl;

    //char*  model_file = "../envs/point_mass.xml";
    //char* mjkey = "../lib/contrib/mjkey.txt";
    std::cout << "N " << n << " STEPS: " << steps << " State dim: " << state_dim << std::endl;

    float* x = (float*) malloc(sizeof(float)*state_dim);
    float* cost = (float*) malloc(sizeof(float)*n);
    float* beta = (float*) malloc(sizeof(float));
    float* nabla = (float*) malloc(sizeof(float));
    float* weight = (float*) malloc(sizeof(float)*n);

    /*
    * The state data stored on host. In this example,
    * the state is only one scalar but is stored on a
    * array with all the following states. Thus we need a int[n]
    * array to the input data.
    */

    /*
    float* h_x = (float*) malloc(sizeof(float)*n*(STEPS+1)*state_dim);
    float* h_u = (float*) malloc(sizeof(float)*STEPS*act_dim);
    float* h_e = (float*) malloc(sizeof(float)*n*STEPS*act_dim);
    float* u_prev = (float*) malloc(sizeof(float)*STEPS*act_dim);

    float* goal = (float*) malloc(sizeof(float)*state_dim);
    float* w = (float*) malloc(sizeof(float)*state_dim);
    // allocate and init and res data.
    goal[0] = 1.0;
    goal[1] = 0.0;
    goal[2] = 0.0;
    goal[3] = 0.0;

    w[0] = 50.0;
    w[1] = 50.0;
    w[2] = 0.25;
    w[3] = 0.25;

    float dt = 0.1;

    //bool test = false;
    bool save = true;
    std::string filename("to_plot.csv");
    PointMassEnv env = PointMassEnv(model_file, mjkey, true);

    PointMassModel* model = new PointMassModel(n, STEPS, dt, state_dim, act_dim, false);
    bool done=false;

    env.get_x(x);

    for (int j=0; j < STEPS; j++){
        h_u[(j*act_dim)+0] = 0.;
        h_u[(j*act_dim)+1] = 0.;
    }
    // send the data on the device.

    float* next_act = (float*) malloc(sizeof(float)*act_dim);

    model->memcpy_set_data(x, h_u, goal, w);


    // run the multiple simulation on the device.
    while(!done){
        model->get_u(u_prev);
        //t1 = std::chrono::system_clock::now();
        model->get_act(next_act);
        //t2 = std::chrono::system_clock::now();
        //fp_ms += t2 - t1;

        done = env.simulate(next_act);
        std::cout << "next_act: " << next_act[0] << ", " << next_act[1] << '\n';
        env.get_x(x);
        model->set_x(x);
    }

    //send act to sim;

    // collect new state;

    // set state in controller;

    // next step


    //delta = fp_ms.count();


    //std::cout << "GPU execution time: " << delta << "ms" << std::endl;

    if(save){
        model->get_inf(h_x, h_u, h_e, cost, beta, nabla, weight);
        to_csv2(filename, h_x, h_u, u_prev, h_e, cost, beta, nabla, weight, n, STEPS, state_dim, act_dim);
    }

    std::cout << "Freeing memory... : " << std::flush;
    free(h_x);
    free(h_u);
    free(h_e);
    free(x);
    free(w);
    free(goal);
    std::cout << "Done" << std::endl;

    delete model;
    hipDeviceReset();

    */
}

void parse_argument (int argc,
                     char const* argv[],
                     std::string& config,
                     std::string& mjkey,
                     std::string& outfile) {
    try {

        TCLAP::CmdLine cmd("Mppi controller", ' ', "0.0");
        TCLAP::ValueArg<std::string> configArg("c",
                                               "config",
                                               "Config file",
                                               false,
                                               "../config/point_mass.yaml",
                                               "string",
                                               cmd);

        TCLAP::ValueArg<std::string> mjkeyArg("k",
                                              "key",
                                              "Mujoco key file",
                                              false,
                                              "../lib/contrib/mjkey.txt",
                                              "string",
                                              cmd);

        TCLAP::ValueArg<std::string> outArg("o",
                                            "out",
                                            "Outpute file",
                                            false,
                                            "to_plot.csv",
                                            "string",
                                            cmd);

        cmd.parse(argc, argv);

        config = configArg.getValue();
        mjkey = mjkeyArg.getValue();
        outfile = outArg.getValue();

    } catch (TCLAP::ArgException &e) {
        std::cerr << "error: " << e.error() << " for arg " << e.argId() << std::endl;
    }
}

void parse_config (std::string& config_file,
                   std::string& model_file,
                   int& samples,
                   int& state_dim,
                   int& act_dim,
                   int& horizon,
                   float& lambda,
                   float** noise,
                   float** init,
                   float** max_a,
                   std::string& cost_type,
                   float** cost_q) {
    float* tmp_noise;
    float* tmp_init;
    float* tmp_max_a;
    float* tmp_cost_q;

    YAML::Node config = YAML::LoadFile(config_file);

    /* env section */
    if (!config["env"])  {
        std::cout << "Please provide a env file in the config file" << std::endl;
        exit(1);
    }
    model_file = config["env"].as<std::string>();

    /* Sample section */
    if (!config["samples"])  {
        std::cout << "Please provide the number of samples in the config file" << std::endl;
        exit(1);
    }
    samples = config["samples"].as<int>();

    /* State section */
    if (!config["state-dim"])  {
        std::cout << "Please provide the state dimension in the config file" << std::endl;
        exit(1);
    }
    state_dim = config["state-dim"].as<int>();

    /* Action section */
    if (!config["action-dim"])  {
        std::cout << "Please provide the action dimension in the config file" << std::endl;
        exit(1);
    }
    act_dim = config["action-dim"].as<int>();

    /* Horizon section */
    if (!config["horizon"])  {
        std::cout << "Please provide the prediction horizon in the config file" << std::endl;
        exit(1);
    }
    horizon = config["horizon"].as<int>();

    /* Lambda section */
    if (!config["lambda"])  {
        std::cout << "Please provide a env file in the config file" << std::endl;
        exit(1);
    }
    lambda = config["lambda"].as<float>();

    /* Noise section */
    {
        if (!config["noise"])  {
            std::cout << "Please provide a noise vector in the config file, should be a array of size action-dim" << std::endl;
            exit(1);
        }
        if (config["noise"].size() != act_dim) {
            std::cout << "Warning: the cost function weights matrix is larger than the action dimension ";
        }
        tmp_noise = (float*) malloc(sizeof(float)*config["max-a"].size());
    }


    /* Init action section */
    {
        if (!config["init-act"])  {
            std::cout << "Please provide a init vector in the config file, should be a array of size action-dim" << std::endl;
            exit(1);
        }
        if (config["init-act"].size() != act_dim) {
            std::cout << "Warning: the cost function weights matrix is larger than the action dimension ";
        }
        tmp_init = (float*) malloc(sizeof(float)*config["max-a"].size());
    }


    /* Max action section */
    {
        if (!config["max-a"])  {
            std::cout << "Please provide a max input vector in the config file, should be a array of size action-dim" << std::endl;
            exit(1);
        }
        if (config["max-a"].size() != act_dim) {
            std::cout << "Warning: the input limit is different than the action dimension " << std::endl;
        }
        tmp_max_a = (float*) malloc(sizeof(float)*config["max-a"].size());
    }
    for (std::size_t i=0; i < config["max-a"].size(); i++) {
        tmp_noise[i] = config["max-a"][i].as<float>();
        tmp_init[i] = config["init-act"][i].as<float>();
        tmp_max_a[i] = config["noise"][i].as<float>();
    }


    /* Cost related section  */
    {
        if (!config["cost"])  {
            std::cout << "Please provide cost function in the config file." << std::endl;
            exit(1);
        }

        if (!config["cost"]["type"]) {
            std::cout << "Please provide cost function type in the config file. Currently supported: quadratic " << std::endl;
            exit(1);
        }
        cost_type = config["cost"]["type"].as<std::string>();

        if (!config["cost"]["w"]) {
            std::cout << "Please provide cost function type in the config file. Currently supported: quadratic " << std::endl;
            exit(1);
        }
        if (config["cost"]["w"].size() != state_dim) {
            std::cout << "Warning: the cost function weights matrix is different than the state dimension " << std::endl;
        }
        tmp_cost_q = (float*) malloc(sizeof(float)*config["cost"]["w"].size());

        for (std::size_t i=0; i< config["cost"]["w"].size(); i++) {
            tmp_cost_q[i] = config["cost"]["w"][i].as<float>();
        }
    }

    *noise = tmp_noise;
    *init = tmp_init;
    *max_a = tmp_max_a;
    *cost_q = tmp_cost_q;

    tmp_noise = nullptr;
    tmp_init = nullptr;
    tmp_max_a = nullptr;
    tmp_cost_q = nullptr;


}
