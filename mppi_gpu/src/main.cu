#include "hip/hip_runtime.h"
#include "point_mass.hpp"
#include "mppi_env.hpp"
#include "mppi_utils.hpp"
#include <iostream>
#include <fstream>

#include <assert.h>

#include <cstdio>
#include <ctime>
#include <chrono>
#include <unistd.h>

#include <tclap/CmdLine.h>
#include <yaml-cpp/yaml.h>
/*
* Same as previous example only this time the parallized
* objects contain a pointer to the data. This will show
* us how to implment a recording of a path for MPPI.
*
* This examples run a simple linear simulation in parallel.
* All this is wrapped in a model class which will later be our
* controller. The Wrapper creates simulation classes on the device.
* It then sets the input data on the classes (can be run in // too).
* finally it runs the simulation in parallel and collects the data
* on the host device.
*/


void to_csv (std::string filename,
            float* x,
            float* u,
            int sample,
            int size,
            int s_dim,
            int a_dim)
{
    std::cout << "Saving data to file...: " << std::flush;
    std::ofstream outfile;
    // create a name for the file output

    outfile.open(filename);

    outfile << "sample" << "," << "x" << "," << "y" << "," << "x_dot" << ","
            << "y_dot" << "," << "u_x" << "," << "u_y" << std::endl;
    for (int i=0; i < sample; i++){
        for (int j=0; j < size; j++){
            outfile << i << ","
                    << x[i*size*s_dim + j*s_dim + 0] << ","
                    << x[i*size*s_dim + j*s_dim + 1] << ","
                    << x[i*size*s_dim + j*s_dim + 2] << ","
                    << x[i*size*s_dim + j*s_dim + 3] << ","
                    << u[i*size*a_dim + j*a_dim + 0] << ","
                    << u[i*size*a_dim + j*a_dim + 1] << std::endl;
        }
    }
    outfile.close();
    std::cout << "Done" << std::endl;
    return;
}

void to_csv2 (std::string filename,
             float* x,
             float* u,
             float* u_prev,
             float* e,
             float* cost,
             float* beta,
             float* nabla,
             float* w,
             int sample,
             int size,
             int s_dim,
             int a_dim) {

    std::cout << "Saving data to file...: " << std::flush;
    std::ofstream outfile;
    // create a name for the file output

    outfile.open(filename);

    outfile << "sample" << "," << "x" << "," << "y" << "," << "x_dot" << ","
            << "y_dot" << "," << "e_x" << "," << "e_y";
    for(int d=0; d < a_dim; d++)
    {
        outfile << "," << "u[" << d << "]";
    }

    for(int d=0; d < a_dim; d++)
    {
        outfile << "," << "u_prev[" << d << "]";
    }

    outfile << "," << "c" <<  "," << "w" << std::endl;

    for (int i=0; i < sample; i++){
        for (int j=0; j < size + 1 ; j++){
            outfile << i << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 0] << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 1] << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 2] << ","
                    << x[i*(size+1)*s_dim + j*s_dim + 3] << ",";
            if (j < size) {
                outfile << e[i*size*a_dim + j*a_dim + 0] << ","
                        << e[i*size*a_dim + j*a_dim + 1];
            } else {
                outfile << ", ";
            }
            // U is of size steps
            if(i < 1 && j < size) {
                outfile << "," << u[j*a_dim + 0] << "," << u[j*a_dim + 1];
                outfile << "," << u_prev[j*a_dim + 0] << "," << u_prev[j*a_dim + 1];
            }else
                outfile << ", , , , ";
            if (i*size + j < sample) {
                outfile << "," << cost[i*size+j] << "," << w[i*size+j];
            }
            outfile << std::endl;
        }
    }


    outfile.close();
    std::cout << "Done" << std::endl;
    return;
}

void parse_argument (int argc,
                     char const* argv[],
                     std::string& config,
                     std::string& mjkey,
                     std::string& outfile);

void parse_config (std::string& configFile,
                   std::string& modelFile,
                   int& samples,
                   int& state_dim,
                   int& act_dim,
                   int& horizon,
                   float& lambda,
                   float** noise,
                   float** init,
                   float** max_a);

int main (int argc, char const* argv[]) {

    std::string configFile;
    std::string mjkeyFile;
    std::string outFile;
    std::string modelFile;

    int n(0);
    int state_dim(0);
    int act_dim(0);
    int steps(0);
    float lambda;
    float* noise;
    float* init;
    float* max_a;

    parse_argument(argc, argv, configFile, mjkeyFile, outFile);

    std::cout << "Config: " << configFile << std::endl;
    std::cout << "MjKey: " << mjkeyFile << std::endl;
    std::cout << "Outfile: " << outFile << std::endl;

    std::cout << max_a << std::endl;
    parse_config(configFile, modelFile, n, state_dim, act_dim, steps, lambda, &noise, &init, &max_a);
    std::cout << max_a << std::endl;

    std::cout << "Parse config output: " << modelFile << " "
              << n << " "
              << state_dim << " "
              << act_dim << " "
              << steps << " "
              << lambda << " " << std::endl;
    std::cout << "max_a: ";
    for (int i = 0; i < act_dim; i++) {
        std::cout << max_a[i] << ' ';
    }
    std::cout << std::endl;

    for (int i = 0; i < act_dim; i++) {
        std::cout << init[i] << ' ';
    }
    std::cout << std::endl;

    for (int i = 0; i < act_dim; i++) {
        std::cout << noise[i] << ' ';
    }
    std::cout << std::endl;
    //char*  modelFile = "../envs/point_mass.xml";
    //char* mjkey = "../lib/contrib/mjkey.txt";


    /*

    int act_dim = 2;
    int state_dim = 4;
    int n = 3000;
    //std::cout << "N " << n << " STEPS: " << STEPS << " State dim: " << state_dim << std::endl;

    float* x = (float*) malloc(sizeof(float)*state_dim);
    float* cost = (float*) malloc(sizeof(float)*n);
    float* beta = (float*) malloc(sizeof(float));
    float* nabla = (float*) malloc(sizeof(float));
    float* weight = (float*) malloc(sizeof(float)*n);

    /*
    * The state data stored on host. In this example,
    * the state is only one scalar but is stored on a
    * array with all the following states. Thus we need a int[n]
    * array to the input data.
    */

    /*
    float* h_x = (float*) malloc(sizeof(float)*n*(STEPS+1)*state_dim);
    float* h_u = (float*) malloc(sizeof(float)*STEPS*act_dim);
    float* h_e = (float*) malloc(sizeof(float)*n*STEPS*act_dim);
    float* u_prev = (float*) malloc(sizeof(float)*STEPS*act_dim);

    float* goal = (float*) malloc(sizeof(float)*state_dim);
    float* w = (float*) malloc(sizeof(float)*state_dim);
    // allocate and init and res data.
    goal[0] = 1.0;
    goal[1] = 0.0;
    goal[2] = 0.0;
    goal[3] = 0.0;

    w[0] = 50.0;
    w[1] = 50.0;
    w[2] = 0.25;
    w[3] = 0.25;

    float dt = 0.1;

    //bool test = false;
    bool save = true;
    std::string filename("to_plot.csv");
    PointMassEnv env = PointMassEnv(modelFile, mjkey, true);

    PointMassModel* model = new PointMassModel(n, STEPS, dt, state_dim, act_dim, false);
    bool done=false;

    env.get_x(x);

    for (int j=0; j < STEPS; j++){
        h_u[(j*act_dim)+0] = 0.;
        h_u[(j*act_dim)+1] = 0.;
    }
    // send the data on the device.

    float* next_act = (float*) malloc(sizeof(float)*act_dim);

    model->memcpy_set_data(x, h_u, goal, w);


    // run the multiple simulation on the device.
    while(!done){
        model->get_u(u_prev);
        //t1 = std::chrono::system_clock::now();
        model->get_act(next_act);
        //t2 = std::chrono::system_clock::now();
        //fp_ms += t2 - t1;

        done = env.simulate(next_act);
        std::cout << "next_act: " << next_act[0] << ", " << next_act[1] << '\n';
        env.get_x(x);
        model->set_x(x);
    }

    //send act to sim;

    // collect new state;

    // set state in controller;

    // next step


    //delta = fp_ms.count();


    //std::cout << "GPU execution time: " << delta << "ms" << std::endl;

    if(save){
        model->get_inf(h_x, h_u, h_e, cost, beta, nabla, weight);
        to_csv2(filename, h_x, h_u, u_prev, h_e, cost, beta, nabla, weight, n, STEPS, state_dim, act_dim);
    }

    std::cout << "Freeing memory... : " << std::flush;
    free(h_x);
    free(h_u);
    free(h_e);
    free(x);
    free(w);
    free(goal);
    std::cout << "Done" << std::endl;

    delete model;
    hipDeviceReset();

    */
}

void parse_argument (int argc,
                     char const* argv[],
                     std::string& config,
                     std::string& mjkey,
                     std::string& outfile) {
    try {

        TCLAP::CmdLine cmd("Mppi controller", ' ', "0.0");
        TCLAP::ValueArg<std::string> configArg("c",
                                               "config",
                                               "Config file",
                                               false,
                                               "../config/point_mass.yaml",
                                               "string",
                                               cmd);

        TCLAP::ValueArg<std::string> mjkeyArg("k",
                                              "key",
                                              "Mujoco key file",
                                              false,
                                              "../lib/contrib/mjkey.txt",
                                              "string",
                                              cmd);

        TCLAP::ValueArg<std::string> outArg("o",
                                            "out",
                                            "Outpute file",
                                            false,
                                            "to_plot.csv",
                                            "string",
                                            cmd);

        cmd.parse(argc, argv);

        config = configArg.getValue();
        mjkey = mjkeyArg.getValue();
        outfile = outArg.getValue();

    } catch (TCLAP::ArgException &e) {
        std::cerr << "error: " << e.error() << " for arg " << e.argId() << std::endl;
    }
}

void parse_config (std::string& configFile,
                   std::string& modelFile,
                   int& samples,
                   int& state_dim,
                   int& act_dim,
                   int& horizon,
                   float& lambda,
                   float** noise,
                   float** init,
                   float** max_a) {
    float* tmp_noise;
    float* tmp_init;
    float* tmp_max_a;
    YAML::Node config = YAML::LoadFile(configFile);

    if (!config["env"])  {
        std::cout << "Please provide a env file in the config file" << std::endl;
        exit(1);
    }
    modelFile = config["env"].as<std::string>();


    if (!config["samples"])  {
        std::cout << "Please provide the number of samples in the config file" << std::endl;
        exit(1);
    }
    samples = config["samples"].as<int>();


    if (!config["state-dim"])  {
        std::cout << "Please provide the state dimension in the config file" << std::endl;
        exit(1);
    }
    state_dim = config["state-dim"].as<int>();


    if (!config["action-dim"])  {
        std::cout << "Please provide the action dimension in the config file" << std::endl;
        exit(1);
    }
    act_dim = config["action-dim"].as<int>();


    if (!config["horizon"])  {
        std::cout << "Please provide the prediction horizon in the config file" << std::endl;
        exit(1);
    }
    horizon = config["horizon"].as<int>();


    if (!config["lambda"])  {
        std::cout << "Please provide a env file in the config file" << std::endl;
        exit(1);
    }
    lambda = config["lambda"].as<float>();


    if (!config["noise"])  {
        std::cout << "Please provide a noise vector in the config file, should be a array of size action-dim" << std::endl;
        exit(1);
    }
    tmp_noise = (float*) malloc(sizeof(float)*act_dim);


    if (!config["init-act"])  {
        std::cout << "Please provide a init vector in the config file, should be a array of size action-dim" << std::endl;
        exit(1);
    }
    tmp_init = (float*) malloc(sizeof(float)*act_dim);


    if (!config["max-a"])  {
        std::cout << "Please provide a max input vector in the config file, should be a array of size action-dim" << std::endl;
        exit(1);
    }
    tmp_max_a = (float*) malloc(sizeof(float)*act_dim);

    for (std::size_t i=0; i < config["max-a"].size(); i++) {
        tmp_noise[i] = config["max-a"][i].as<float>();
        tmp_init[i] = config["init-act"][i].as<float>();
        tmp_max_a[i] = config["noise"][i].as<float>();
    }
    *noise = tmp_noise;
    *init = tmp_init;
    *max_a = tmp_max_a;
    std::cout << max_a << std::endl;

}
