#include "hip/hip_runtime.h"
#include "point_mass.hpp"
#include <iostream>
#include <fstream>

#include <assert.h>

#include <cstdio>
#include <ctime>
#include <chrono>
#include <unistd.h>

/*
 * Same as previous example only this time the parallized
 * objects contain a pointer to the data. This will show
 * us how to implment a recording of a path for MPPI.
 *
 * This examples run a simple linear simulation in parallel.
 * All this is wrapped in a model class which will later be our
 * controller. The Wrapper creates simulation classes on the device.
 * It then sets the input data on the classes (can be run in // too).
 * finally it runs the simulation in parallel and collects the data
 * on the host device.
 */

 #define TOL 1e-6

void to_csv(float* x, size_t size, size_t sample){
  std::ofstream outfile;
  // create a name for the file output
  std::string filename = "exampleOutput.csv";

  outfile.open(filename);

  outfile << "sample" << "," << "x" << "," << "y" << "," << "x_dot" << "," << "y_dot" << std::endl;
  for (int j=0; j < sample; j++){
    for (int i=0; i < size; i++){
      outfile << j << "," << x[i*4 + 0] << "," << x[i*4 + 1] << "," << x[i*4 + 2] << "," << x[i*4 + 3] << std::endl;
    }
  }
  outfile.close();
}

int main(){


  std::chrono::time_point<std::chrono::system_clock> t1;
  std::chrono::time_point<std::chrono::system_clock> t2;
  std::chrono::duration<double, std::milli> fp_ms;
  double delta;

  int act_dim = 2;
  int state_dim = 4;

  int n = 1;

  float dt = 1.;

  /*
   * copy of our models on host. Should ultimatly
   * be removed and the models object should be stored on
   * device only.
   */
  PointMassModelGpu* models = new PointMassModelGpu[n];
  /*
   * Model Gpu wrapper, this will allow to offer one entry
   * interface to the controller that is not device or host specific.
   */
  PointMassModel model = PointMassModel(n, STEPS, dt);
  /*
   * The state data stored on host. In this example,
   * the state is only one scalar but is stored on a
   * array with all the following states. Thus we need a int[n]
   * array to the input data.
   */

  float* h_x;
  float* h_u;

  /*
   * This variable stores the output result. In this
   * example it is a n*steps array but it will be continous
   * in device memory. so a int* array will be easier to work with.
   */

  float* h_o;


  // allocate and init and res data.
  h_x = (float*) malloc(sizeof(float)*n*state_dim);
  h_u = (float*) malloc(sizeof(float)*n*STEPS*act_dim);

  h_o = (float*) malloc(sizeof(float)*n*STEPS*state_dim);
  for (int i=0; i < n; i++){
    h_x[i*state_dim+0] = 0.;
    h_x[i*state_dim+1] = 0.;
    h_x[i*state_dim+2] = 0.;
    h_x[i*state_dim+3] = 0.;
    for (int j=0; j < STEPS; j++){
      h_u[(i*STEPS*act_dim)+(j*act_dim)+0] = 0.01;
      h_u[(i*STEPS*act_dim)+(j*act_dim)+1] = 0.01;
    }
  }
  // send the data on the device.
  model.memcpy_set_data(h_x, h_u);

  t1 = std::chrono::system_clock::now();

  // run the multiple simulation on the device.
  model.sim();


  t2 = std::chrono::system_clock::now();
  fp_ms = t2 - t1;
  delta = fp_ms.count();

  std::cout << "GPU execution time: " << delta << "ms" << std::endl;

  // get the data from the device.
  model.memcpy_get_data(h_o);
  //std::cout << "Saving data to file...: ";
  //to_csv(h_o, STEPS, n);
  //std::cout << "Done" << std::endl;

  {
    float** x;
    float* u;
    float x_gain[state_dim];
    float u_gain[act_dim];

    u_gain[0] = dt*dt/2.0;
    u_gain[1] = dt;
    x_gain[0] = 1;
    x_gain[1] = dt;
    x_gain[2] = 0;
    x_gain[3] = 1;

    x = (float**) malloc(sizeof(float*)*n);
    u = (float*) malloc(sizeof(float)*STEPS*act_dim);
    for(int i=0; i < STEPS; i++){
      u[i*act_dim + 0] = 0.01;
      u[i*act_dim + 1] = 0.01;
    }

    for(int i=0; i < n; i++){
      x[i] = (float*) malloc(sizeof(float)*STEPS*state_dim);
      x[i][0] = 0.0;
      x[i][1] = 0.0;
      x[i][2] = 0.0;
      x[i][3] = 0.0;
      models[i].init(x[i], 0, u, x_gain, state_dim, u_gain, act_dim);
    }

    t1 = std::chrono::system_clock::now();

    // run the same code for on the cpu to evaluate the improvement.
    for(int i=0; i < n; i++){
      models[i].run();
    }

    t2 = std::chrono::system_clock::now();
    fp_ms = t2 - t1;
    delta = fp_ms.count();

    std::cout << "Sequencial execution time: " << delta << "ms" << std::endl;


    // free the memory.
    for(int i=0; i<n; i++){
      for (int j=0; j<STEPS; j++){
        for (int k=0; k<state_dim; k++){
          std::cout << "h[" << i*STEPS*state_dim + j*state_dim + k << "]: " << h_o[i*STEPS*state_dim + j*state_dim + k]
                    << " x["<< i << "][" << j*state_dim + k << "]: " << x[i][j*state_dim + k]
                    << " diff: " << h_o[i*STEPS*state_dim + j*state_dim + k] - x[i][j*state_dim + k] << std::endl;
          //assert(fabs(h_o[i*STEPS*state_dim + j*state_dim + k] - x[i][j*state_dim + k]) < TOL );
        }
        std::cout << std::endl;
      }
      std::cout << std::endl;
    }
    std::cout << "Test passed" << std::endl;
  }

  std::cout << "Freeing memory... : ";
  free(h_x);
  free(h_o);
  free(h_u);
  std::cout << "Done" << std::endl;
}
